#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include "cu_errchk.h"

// red = even row, even col
// blue = odd row, odd col
// green1 = even row, odd col green
// green2 = odd row, evel col green
typedef enum { RED = 0, GREEN1 = 1, GREEN2 = 2, BLUE = 3 } color_t;

__device__ __inline__ color_t fc(int row, int col) {
    // even row + odd col = blue, odd row + even col = red. match = green.
    return (color_t) (((row & 1) << 1) + (col & 1));
}
/* R G R G 
 * G B G B 
 * R G R G 
 * G B G B 
 */
__global__ void linear_demosaic_r(ushort *raw_in, int width, int height,
                                  ushort4 *out) {
    int row = (threadIdx.y + blockIdx.y * blockDim.y) * 2;
    int col = (threadIdx.x + blockIdx.x * blockDim.x) * 2;
    
    if (row < 1 || col < 1 || row >= (height - 1) || col >= (width - 1))
        return;

    // shared buffers: uncomment and use this when the whole thing works without it.
    /*
    int local_row = threadIdx.y * 2 + 1;
    int local_col = threadIdx.x * 2;
    int local_width = 2 * blockDim.x;
    int local_height = 2 * blockDim.y;*/
    /* fill a shared buffer of raw values.
     * R G
     * G B
     * we have a thread for each red, so each thread fills in its own element,
     * and the ones above it and to its right.
     */
    /*
    __shared__ ushort in_buf[4 * blockDim.x * blockDim.y];
    in_buf[(local_row - 1) * local_width + local_col] =
        raw_in[(row - 1) * width + col];
    in_buf[(local_row - 1) * local_width + (local_col + 1)] =
        raw_in[(row - 1) * width + (col + 1)];
    in_buf[local_row * local_width + local_col] = raw_in[row * width + col];
    in_buf[local_row * local_width + (local_col + 1)] =
        raw_in[row * width + (col + 1)];
    */
    
    ushort4 px;
    px.x = raw_in[row * width + col]; //red: this pixel
    // green: average of pixels directly adjacent to us
    px.y = (raw_in[(row - 1) * width + col] +
            raw_in[(row + 1) * width + col] +
            raw_in[row * width + (col - 1)] +
            raw_in[row * width + (col + 1)])
           / 4;
    // blue: average of pixels diagonally adjacent to us
    px.z = (raw_in[(row - 1) * width + (col - 1)] +
            raw_in[(row - 1) * width + (col + 1)] +
            raw_in[(row + 1) * width + (col - 1)] +
            raw_in[(row + 1) * width + (col + 1)])
           / 4;
    //printf("(%d, %d) r:  (%d, %d, %d)\n", row, col, px.x, px.y, px.z);
    out[row * width + col] = px;
}


__global__ void linear_demosaic_g1(ushort *raw_in, int width, int height,
                                  ushort4 *out) {
    int row = (blockIdx.y * blockDim.y + threadIdx.y) * 2;
    int col = (blockIdx.x * blockDim.x + threadIdx.x) * 2 + 1;

    if (row < 1 || col < 1 || row >= (height - 1) || col >= (width - 1))
        return;
    
    /*
    int local_row = threadIdx.y;
    int local_col = threadIdx.x * 2 + (row & 1);
    int local_width = 2 * blockDim.x;
    int local_height = blockDim.y;
    __shared__ ushort in_buf[2 * blockDim.x * blockDim.y];
    */  

    ushort4 px;
    // red: left and right average
    px.x = (raw_in[row * width + (col - 1)] +
            raw_in[row * width + (col + 1)])
        / 2;
    // green: this pixel
    px.y = raw_in[row * width + col];
    // blue: above and below average
    px.z = (raw_in[(row - 1) * width + col] +
            raw_in[(row + 1) * width + col])
        / 2;
    //printf("(%d, %d) g1: (%d, %d, %d)\n", row, col, px.x, px.y, px.z);
    out[row * width + col] = px;
}

__global__ void linear_demosaic_g2(ushort *raw_in, int width, int height,
                                  ushort4 *out) {
    int row = (blockIdx.y * blockDim.y + threadIdx.y) * 2 + 1;
    int col = (blockIdx.x * blockDim.x + threadIdx.x) * 2;

    if (row < 1 || col < 1 || row >= (height - 1) || col >= (width - 1))
        return;
    
    /*
    int local_row = threadIdx.y;
    int local_col = threadIdx.x * 2 + (row & 1);
    int local_width = 2 * blockDim.x;
    int local_height = blockDim.y;
    __shared__ ushort in_buf[2 * blockDim.x * blockDim.y];
    */  

    ushort4 px;

    // red: above and below average
    px.x = (raw_in[(row - 1) * width + col] +
            raw_in[(row + 1) * width + col])
        / 2;
    // green: this pixel
    px.y = raw_in[row * width + col];
    // blue: left and right average
    px.z = (raw_in[row * width + (col - 1)] +
            raw_in[row * width + (col + 1)])
        / 2;
    //printf("(%d, %d) g2: (%d, %d, %d)\n", row, col, px.x, px.y, px.z);
    out[row * width + col] = px;
}

__global__ void linear_demosaic_b(ushort *raw_in, int width, int height,
                                  ushort4 *out) {
    int row = (blockIdx.y * blockDim.y + threadIdx.y) * 2 + 1;
    int col = (blockIdx.x * blockDim.x + threadIdx.x) * 2 + 1;

    if (row < 1 || col < 1 || row >= (height - 1) || col >= (width - 1))
        return;

    ushort4 px;

    // red: average of pixels diagonally adjacent to us
    px.x = (raw_in[(row - 1) * width + (col - 1)] +
            raw_in[(row - 1) * width + (col + 1)] +
            raw_in[(row + 1) * width + (col - 1)] +
            raw_in[(row + 1) * width + (col + 1)])
          / 4;
    // green: average of pixels directly adjacent to us
    px.y = (raw_in[(row - 1) * width + col] +
            raw_in[(row + 1) * width + col] +
            raw_in[row * width + (col - 1)] +
            raw_in[row * width + (col + 1)])
          / 4;
    px.z = raw_in[row * width + col]; //blue: this pixel
    //printf("(%d, %d) b:  (%d, %d, %d)\n", row, col, px.x, px.y, px.z);
    out[row * width + col] = px;
}

inline int updiv(int n, int d) {
    return (n + d - 1) / d;
}

hipStream_t colorStreams[4];
void initStreams() {
    for (int i = 0; i < 4; i++) {
        hipStreamCreate(&colorStreams[i]);
    }
}

void linear_demosaic_cu(ushort *raw_in, ushort *img_out, int width, int height) {
    ushort *deviceIn;
    ushort4 *deviceOut;

    // initialize buffers
    hipMalloc(&deviceIn, sizeof(ushort) * width * height);
    hipMalloc(&deviceOut, sizeof(ushort4) * width * height);
    hipMemcpy(deviceIn, raw_in, sizeof(ushort) * width * height,
	       hipMemcpyHostToDevice);

    // launch kernels
    const int blkSide = 16;
    dim3 blkDim(blkSide, blkSide);
    dim3 gridDim(updiv(width, blkSide * 2), updiv(height, blkSide * 2));

    //printf("launch? %d %d %d %d\n", blkDim.x, blkDim.y, gridDim.x, gridDim.y);
    linear_demosaic_r<<<gridDim, blkDim, 0, colorStreams[RED]>>>(deviceIn, width, height, deviceOut);
    //gpuErrchk( hipPeekAtLastError() );
    linear_demosaic_g1<<<gridDim, blkDim, 0, colorStreams[GREEN1]>>>(deviceIn, width, height, deviceOut);
    //gpuErrchk( hipPeekAtLastError() );
    linear_demosaic_g2<<<gridDim, blkDim, 0, colorStreams[GREEN2]>>>(deviceIn, width, height, deviceOut);
    //gpuErrchk( hipPeekAtLastError() );
    linear_demosaic_b<<<gridDim, blkDim, 0, colorStreams[BLUE]>>>(deviceIn, width, height, deviceOut);
    //gpuErrchk( hipPeekAtLastError() );
    //gpuErrchk( hipDeviceSynchronize() );
    hipDeviceSynchronize();

    // copy results & free buffers
    hipMemcpy(img_out, deviceOut, sizeof(ushort4) * width * height,
               hipMemcpyDeviceToHost);
    hipFree(deviceIn);
    hipFree(deviceOut);
}